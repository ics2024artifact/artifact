#include "hip/hip_runtime.h"
#include "fused_distance_nn/l2_exp.cuh"
#include "fused_distance_nn/cutlass_base_customized.cuh"
#include "header.cuh"
#include <hip/hip_runtime.h>
#include <iostream>

using namespace my_test;
// to be modified!!!!!
int main(){
    using IdxT = int;
    using DataT = float;
    using L2Op                  = l2_exp_cutlass_op<DataT, DataT>;
    using OutT = KeyValuePair<IdxT, DataT>;
    using kvp_cg_min_reduce_op_ = kvp_cg_min_reduce_op<DataT, IdxT, OutT>;
    using MinAndDistanceReduceOp = MinAndDistanceReduceOpImpl<IdxT, DataT>;
    using KVPMinReduce = KVPMinReduceImpl<IdxT, DataT>;

    OutT* min, *dmin;
    DataT* x, *dx;
    DataT* y, *dy;
    DataT* xn, *dxn;
    DataT* yn, *dyn;
    IdxT m = 32768;
    IdxT n = 128;
    IdxT k = 128;
    int* workspace, *dworkspace;
    MinAndDistanceReduceOp redOp;
    KVPMinReduce pairRedOp;
    bool sqrt = false;
    hipStream_t stream;
    hipStreamCreate ( &stream) ;

 
    int lda, ldb, ldd;
    lda = k, ldb = k, ldd = n;
    kvp_cg_min_reduce_op_ cg_reduce_op;
    L2Op L2_dist_op(sqrt);

    hipMalloc((void**)&dx, sizeof(DataT) * m * k);
    hipMalloc((void**)&dy, sizeof(DataT) * n * k);
    hipMalloc((void**)&dxn, sizeof(DataT) * m);
    hipMalloc((void**)&dyn, sizeof(DataT) * n);
    hipMalloc((void**)&dmin, sizeof(OutT) * m);
    hipMalloc((void**)&dworkspace, sizeof(IdxT));
    // Sample
    x = (DataT*)malloc(sizeof(DataT) * m * k);
    // Centroids
    y = (DataT*)malloc(sizeof(DataT) * n * k);
    // X-norm
    xn = (DataT*)malloc(sizeof(DataT) * m);
    // Y-norm
    yn = (DataT*)malloc(sizeof(DataT) * n);
    // Best-fit Centroids
    min = (OutT*)malloc(sizeof(OutT) * m); 
    workspace = (IdxT*)malloc(sizeof(IdxT));
    *workspace = 0;
    
    for(int i = 0; i < m * k / 2; ++i)     x[i] = 1;
    for(int i = 0; i < n * k / 2; ++i)     y[i] = 1; 
    for(int i = m * k / 2; i < m * k; ++i) x[i] = 2;
    for(int i = n * k / 2; i < n * k; ++i) y[i] = 2; 
    for(int i = 0; i < m / 2; ++i)     xn[i] = 1 * k;
    for(int i = 0; i < n / 2; ++i)     yn[i] = 1 * k; 
    for(int i = m / 2; i < m; ++i) xn[i] = 4 * k;
    for(int i = n / 2; i < n; ++i) yn[i] = 4 * k; 
    for(int i = 0; i < m; ++i) min[i].value = std::numeric_limits<float>::max(); 
    for(int i = 0; i < m; ++i) min[i].key = -1; 

    // for(int i = 0; i < m; ++i) printf("xn, id = %d, norm = %f\n", i, xn[i]);
    // for(int i = 0; i < n; ++i) printf("yn, id = %d, norm = %f\n", i, yn[i]);
    // for(int i = 0; i < m * k; ++i) printf("x, id = %d, val = %f\n", int(i / k), x[i]);
    // for(int i = 0; i < n * k; ++i) printf("y, id = %d, val = %f\n", int(i / k), y[i]);
  
    hipMemcpy((void*)dx, (void*)x, sizeof(DataT) * m * k, hipMemcpyHostToDevice);
    hipMemcpy((void*)dy, (void*)y, sizeof(DataT) * n * k, hipMemcpyHostToDevice);
    hipMemcpy((void*)dxn, (void*)xn, sizeof(DataT) * m, hipMemcpyHostToDevice);
    hipMemcpy((void*)dyn, (void*)yn, sizeof(DataT) * n, hipMemcpyHostToDevice);
    hipMemcpy((void*)dmin, (void*)min, sizeof(OutT) * m, hipMemcpyHostToDevice);
    hipMemcpy((void*)dworkspace, (void*)workspace, sizeof(IdxT), hipMemcpyHostToDevice);
  
    hipDeviceSynchronize();
				hipEvent_t beg, end;
    			hipEventCreate(&beg);
    			hipEventCreate(&end);
    			float elapsed;
		hipEventRecord(beg);
    cutlassFusedDistanceNN_codegen<DataT,
                           DataT,
                           OutT,
                           IdxT,
                           16 / sizeof(DataT),
                           kvp_cg_min_reduce_op_,
                           L2Op, 
                           MinAndDistanceReduceOp,
                           KVPMinReduce>(dx,
                                         dy,
                                         dxn,
                                         dyn,
                                         m, 
                                         n,
                                         k, 
                                         lda,
                                         ldb,
                                         ldd,
                                         dmin,
                                         dworkspace,
                                         cg_reduce_op,
                                         L2_dist_op,
                                         redOp,
                                         pairRedOp,
                                         stream);
  hipDeviceSynchronize();
	hipEventRecord(end);
  hipEventSynchronize(beg);
  hipEventSynchronize(end);
  hipEventElapsedTime(&elapsed, beg, end);
  double gflops = (double(2 * 1 * double(m) * double(n) * double(k)) / (1e9)) / (elapsed / 1e3);
  //printf("%d, %d, %d, %f, %f\n", m, n, k, elapsed, gflops);
  hipMemcpy((void*)x, (void*)dx, sizeof(DataT) * m * k, hipMemcpyDeviceToHost);
  hipMemcpy((void*)y, (void*)dy, sizeof(DataT) * n * k, hipMemcpyDeviceToHost);
  hipMemcpy((void*)xn, (void*)dxn, sizeof(DataT) * m, hipMemcpyDeviceToHost);
  hipMemcpy((void*)yn, (void*)dyn, sizeof(DataT) * n, hipMemcpyDeviceToHost);
  hipMemcpy((void*)min, (void*)dmin, sizeof(OutT) * m, hipMemcpyDeviceToHost);

  // for(int i = 0; i < m; ++i){
  //   printf("id=%d, class_id = %d, dis = %f\n", i, min[i].key, min[i].value);
  // }
  for (int i = 0; i < m/2; i++)
    if (min[i].value > 0.1 ) {
      printf("Wrong!");
      return 0;
    }

    // hipError_t hipError_t = hipGetLastError();
    // if (hipError_t != hipSuccess) {
    //     std::cerr << "CUDA Error (hipMalloc): " << hipGetErrorString(hipError_t) << std::endl;
    //     return 1;
    // }
  
  // update_centroids

  // for(int i = 0; i < m; ++i) printf("xn, id = %d, norm = %f\n", i, xn[i]);
  // for(int i = 0; i < n; ++i) printf("yn, id = %d, norm = %f\n", i, yn[i]);
  // for(int i = 0; i < m * k; ++i) printf("x, id = %d, val = %f\n", i / k, x[i]);
  // for(int i = 0; i < n * k; ++i) printf("y, id = %d, val = %f\n", i / k, y[i]);
  return 0;
}